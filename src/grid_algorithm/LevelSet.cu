#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
// Level set
// Copyright (c) (2018-), Bo Zhu, Xingyu Ni
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////
#include <numeric>
#include <set>
#include <queue>
#include <utility>
#include <iostream>
#include "LevelSet.h"
#include "Constants.h"
#include "Timer.h"

namespace Meso {

	template<int d> LevelSet<d>::LevelSet(const Grid<d> _grid)
	{
		Init(_grid);
	}

	template<int d> void LevelSet<d>::Init(const Grid<d> _grid)
	{
		phi.Init(_grid, std::numeric_limits<real>::max());
	}


	//template<int d> real LevelSet<d>::Phi(const VectorD& pos) const
	//{
	//	return intp->Interpolate_Centers(phi, pos);
	//}

	//template<int d> real LevelSet<d>::Curvature(const VectorD& pos) const
	//{
	//	real one_over_dx = (real)1 / grid.dx; real one_over_two_dx = (real).5 * one_over_dx; real curvature = (real)0;
	//	for (int i = 0; i < d; i++) {
	//		VectorD normal_left = Normal(pos - VectorD::Unit(i) * grid.dx);
	//		VectorD normal_right = Normal(pos + VectorD::Unit(i) * grid.dx);
	//		curvature += (normal_right[i] - normal_left[i]) * one_over_two_dx;
	//	}
	//	return abs(curvature) < one_over_dx ? curvature : (curvature <= (real)0 ? (real)-1 : (real)1) * one_over_dx;

	//}

	//template<int d> Vector<real, d> LevelSet<d>::Closest_Point(const VectorD& pos, const real epsilon) const
	//{
	//	VectorD normal = Gradient(pos); normal.normalize();
	//	return pos - normal * (Phi(pos) + epsilon);
	//}

	//template<int d> Vector<real, d> LevelSet<d>::Closest_Point_With_Iterations(const VectorD& pos, const int max_iter/*=5*/) const
	//{
	//	VectorD intf_pos = pos;
	//	for (int i = 0; i < max_iter; i++) {
	//		intf_pos = Closest_Point(intf_pos);
	//		if (Phi(intf_pos) < (real)0)return intf_pos;
	//	}
	//	return intf_pos;
	//}

	template<int d> real LevelSet<d>::Cell_Fraction(const VectorDi& cell) const
	{
		real dx = phi.grid.dx;
		return (real).5 - MathFunc::Clamp(phi(cell), -(real).5 * dx, (real).5 * dx) / dx;
	}

	//////////////////////////////////////////////////////////////////////////
	////Fast marching method

	template<int d> void LevelSet<d>::Fast_Marching(const real band_width)
	{
		Grid<d> grid = phi.grid;
		//Timer timer;
		//timer.Reset();
		

		Field<real, d> tent(grid, band_width < 0 ? std::numeric_limits<real>::max() : band_width);
		Array<ushort> done(grid.DoF(), 0);
		using PRI = std::pair<real, int>;
		std::priority_queue<PRI, Array<PRI>, std::greater<PRI> > heaps[2];
		const int cell_num = grid.DoF();
		//real far_from_intf_phi_val=grid.dx*(real)5;

		//////////////////////////////////////////////////////////////////////////
		////precondition
		//// find interface cells
#pragma omp parallel for
		for (int i = 0; i < cell_num; i++) {
			const VectorDi cell = grid.Coord(i);
			//if(abs(phi(cell))>far_from_intf_phi_val)continue;		////ATTENTION: this might cause problem if the levelset is badly initialized

			for (int j = 0; j < Grid<d>::Neighbor_Node_Number(); j++) {
				VectorDi nb = grid.Neighbor_Node(cell, j);
				if (!grid.Valid(nb))continue;
				if (Is_Interface(cell, nb)) {
					done[i] = 1; break;
				}
			}
		}
		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM Precond: find interface");

		//// calculate interface phi values
#pragma omp parallel for
		for (int c = 0; c < cell_num; c++) {
			if (!done[c])continue;		////select interface cells
			const VectorDi cell = grid.Coord(c);

			VectorD correct_phi = VectorD::Ones() * std::numeric_limits<real>::max();
			VectorDi correct_axis = VectorDi::Zero();
			for (int i = 0; i < Grid<d>::Neighbor_Node_Number(); i++) {
				VectorDi nb = grid.Neighbor_Node(cell, i);
				if (!grid.Valid(nb)) continue;
				const int nb_idx = grid.Index(nb);
				if (done[nb_idx] && Is_Interface(cell, nb)) {
					real c_phi = Theta(phi(cell), phi(nb)) * grid.dx; // always non-negative
					int axis = grid.Neighbor_Node_Axis(i);
					correct_axis[axis] = 1;
					correct_phi[axis] = std::min(correct_phi[axis], c_phi);
				}
			}
			if (correct_axis != VectorDi::Zero()) {
				real hmnc_mean = (real)0;
				for (int i = 0; i < d; i++) {
					if (correct_axis[i] == 0)continue;
					hmnc_mean += (real)1 / (correct_phi[i] * correct_phi[i]);
				}
				hmnc_mean = sqrt((real)1 / hmnc_mean);
				tent(cell) = hmnc_mean;
			}
			else {
				Error("[Levelset] bad preconditioning");
			}
		}

		//// initialize heap with front cells
#pragma omp parallel for
		for (int i = 0; i < cell_num; i++) {
			const VectorDi cell = grid.Coord(i);
			if (!done[i]) continue;
			bool is_relaxed = false;
			for (int j = 0; j < Grid<d>::Neighbor_Node_Number(); j++) {
				VectorDi nb = grid.Neighbor_Node(cell, j);
				if (!grid.Valid(nb))continue;
				const int nb_idx = grid.Index(nb);
				if (done[nb_idx]) { is_relaxed = true; break; }
			}

			if (is_relaxed) {
				real temp = Solve_Eikonal(cell, tent, done);
				if (temp < tent(cell)) {
					tent(cell) = temp;
				}
			}
#pragma omp critical
			{heaps[MathFunc::Sign(tent(cell)) > 0 ? 0 : 1].push(PRI(tent(cell), i)); }
		}

		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM: Build heap");

		//// heap traversing
#pragma omp parallel for
		for (int h = 0; h < 2; h++) {
			auto& heap = heaps[h];
			while (!heap.empty()) {
				const real top_val = heap.top().first;
				const int cell_idx = heap.top().second;
				const VectorDi cell = grid.Coord(cell_idx);
				heap.pop();
				if (tent(cell) != top_val) continue;
				done[cell_idx] = 1;

				for (int i = 0; i < Grid<d>::Neighbor_Node_Number(); i++) {
					VectorDi nb = grid.Neighbor_Node(cell, i);
					if (!grid.Valid(nb))continue;
					const int nb_idx = grid.Index(nb);
					//relaxation
					if (!done[nb_idx]) {
						real temp = Solve_Eikonal(nb, tent, done);
#pragma omp critical
						{
							if (temp < tent(nb)) { tent(nb) = temp; heap.push(PRI(temp, nb_idx)); }
						}
					}
				}
			}
		}

		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM: Traverse heap");

		ArrayFunc::Binary_Transform(
			phi.Data(),
			tent.Data(),
			[=](const real phi_i, const real tent_i) {return MathFunc::Sign(phi_i) * tent_i; },
			phi.Data()
		);
	}

	template<int d> bool LevelSet<d>::Solve_Quadratic(const real p1, const real p2, const real dx, real& rst)
	{
		if (abs(p1) >= abs(p2) + dx) { rst = p2 + dx; return true; }
		else if (abs(p2) >= abs(p1) + dx) { rst = p1 + dx; return true; }
		else {
			real delta = (real)2 * dx * dx - pow(p1 - p2, 2);
			if (delta < (real)0) { std::cerr << "Error: [Levelset] negative delta in Solve_Quadratic_2" << std::endl; return false; }
			rst = (real).5 * (p1 + p2 + sqrt(delta)); return true;
		}
	}

	template<int d> bool LevelSet<d>::Solve_Quadratic(const real p1, const real p2, const real p3, const real dx, real& rst)
	{
		real delta = pow(p1 + p2 + p3, 2) - (real)3 * (p1 * p1 + p2 * p2 + p3 * p3 - dx * dx);
		if (delta < (real)0) {
			int i = 0; real p_max = abs(p1); if (abs(p2) > p_max) { i = 1; p_max = abs(p2); }if (abs(p3) > p_max) { i = 2; p_max = abs(p3); }
			real q1, q2; if (i == 0) { q1 = p2; q2 = p3; }
			else if (i == 1) { q1 = p1; q2 = p3; }
			else { q1 = p1; q2 = p2; }
			return Solve_Quadratic(q1, q2, dx, rst);
		}
		rst = (p1 + p2 + p3 + sqrt(delta))/3.0; 
		return true;
	}

	template<int d> real LevelSet<d>::Solve_Eikonal(const VectorDi& cell, const Field<real, d>& tent, const Array<ushort>& done)
	{
		const Grid<d> grid = phi.grid;

		// calculate correct phi from nb interface cells
		VectorD correct_phi = VectorD::Ones() * std::numeric_limits<real>::max();
		VectorDi correct_axis = VectorDi::Zero();
		for (int i = 0; i < Grid<d>::Neighbor_Node_Number(); i++) {
			VectorDi nb = grid.Neighbor_Node(cell, i);
			if (!grid.Valid(nb)) continue;
			const int nb_idx = grid.Index(nb);
			if (done[nb_idx]) {
				int axis = grid.Neighbor_Node_Axis(i); correct_axis[axis] = 1;
				correct_phi[axis] = std::min(correct_phi[axis], tent(nb));
			}
		}
		// update phi on the cell
		real new_phi;
		int n = correct_axis.sum();

		switch (n) {
		case 1: {
			real c_phi;
			for (int i = 0; i < d; i++)
				if (correct_axis[i] != 0) { c_phi = correct_phi[i]; break; }
			new_phi = grid.dx + c_phi;
		} break;
		case 2: {
			real p[2];
			int j = 0;
			for (int i = 0; i < d; i++)
				if (correct_axis[i] != 0) p[j++] = correct_phi[i];
			Solve_Quadratic(p[0], p[1], grid.dx, new_phi);
		} break;
		case 3: {
			Solve_Quadratic(correct_phi[0], correct_phi[1], correct_phi[2], grid.dx, new_phi);
		} break;
		default: {
			Error("[Levelset] bad solving Eikonal");
		} break;
		}
		return new_phi;
	}

	template class LevelSet<2>;
	template class LevelSet<3>;

}