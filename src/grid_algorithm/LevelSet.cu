#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
// Level set
// Copyright (c) (2018-), Bo Zhu, Xingyu Ni
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////
#include <numeric>
#include <set>
#include <queue>
#include <utility>
#include <iostream>
#include "LevelSet.h"
#include "Constants.h"
#include "Timer.h"

namespace Meso {

	real Solve_Upwind_Eikonal2(const real p1, const real p2, const real dx)
	{
		if (abs(p1) >= abs(p2) + dx)  return p2 + dx;
		else if (abs(p2) >= abs(p1) + dx)  return p1 + dx;
		else {
			real delta = (real)2 * dx * dx - pow(p1 - p2, 2);
			Assert(delta >= 0, "Error: solve quadratic delta={}", delta);
			return (real).5 * (p1 + p2 + sqrt(delta));
		}
	}

	real Solve_Upwind_Eikonal3(const real p1, const real p2, const real p3, const real dx)
	{
		real delta = pow(p1 + p2 + p3, 2) - (real)3 * (p1 * p1 + p2 * p2 + p3 * p3 - dx * dx);
		if (delta < (real)0) {
			int i = 0; real p_max = abs(p1); if (abs(p2) > p_max) { i = 1; p_max = abs(p2); }if (abs(p3) > p_max) { i = 2; p_max = abs(p3); }
			real q1, q2; if (i == 0) { q1 = p2; q2 = p3; }
			else if (i == 1) { q1 = p1; q2 = p3; }
			else { q1 = p1; q2 = p2; }
			return Solve_Upwind_Eikonal2(q1, q2, dx);
		}
		return (p1 + p2 + p3 + sqrt(delta)) / 3.0;
	}

	template<int d> LevelSet<d>::LevelSet(const Grid<d> _grid)
	{
		Init(_grid);
	}

	template<int d> void LevelSet<d>::Init(const Grid<d> _grid)
	{
		phi.Init(_grid, std::numeric_limits<real>::max());
	}


	//template<int d> real LevelSet<d>::Phi(const VectorD& pos) const
	//{
	//	return intp->Interpolate_Centers(phi, pos);
	//}

	//template<int d> real LevelSet<d>::Curvature(const VectorD& pos) const
	//{
	//	real one_over_dx = (real)1 / grid.dx; real one_over_two_dx = (real).5 * one_over_dx; real curvature = (real)0;
	//	for (int i = 0; i < d; i++) {
	//		VectorD normal_left = Normal(pos - VectorD::Unit(i) * grid.dx);
	//		VectorD normal_right = Normal(pos + VectorD::Unit(i) * grid.dx);
	//		curvature += (normal_right[i] - normal_left[i]) * one_over_two_dx;
	//	}
	//	return abs(curvature) < one_over_dx ? curvature : (curvature <= (real)0 ? (real)-1 : (real)1) * one_over_dx;

	//}

	//template<int d> Vector<real, d> LevelSet<d>::Closest_Point(const VectorD& pos, const real epsilon) const
	//{
	//	VectorD normal = Gradient(pos); normal.normalize();
	//	return pos - normal * (Phi(pos) + epsilon);
	//}

	//template<int d> Vector<real, d> LevelSet<d>::Closest_Point_With_Iterations(const VectorD& pos, const int max_iter/*=5*/) const
	//{
	//	VectorD intf_pos = pos;
	//	for (int i = 0; i < max_iter; i++) {
	//		intf_pos = Closest_Point(intf_pos);
	//		if (Phi(intf_pos) < (real)0)return intf_pos;
	//	}
	//	return intf_pos;
	//}

	template<int d> real LevelSet<d>::Cell_Fraction(const VectorDi& cell) const
	{
		real dx = phi.grid.dx;
		return (real).5 - MathFunc::Clamp(phi(cell), -(real).5 * dx, (real).5 * dx) / dx;
	}

	//////////////////////////////////////////////////////////////////////////
	////Fast marching method

	template<int d> void LevelSet<d>::Fast_Marching(const real band_width)
	{
		Grid<d> grid = phi.grid;
		//Timer timer;
		//timer.Reset();
		

		Field<real, d> tent(grid, band_width < 0 ? std::numeric_limits<real>::max() : band_width);
		Array<ushort> done(grid.DoF(), 0);
		using PRI = std::pair<real, int>;
		std::priority_queue<PRI, Array<PRI>, std::greater<PRI> > heaps[2];
		const int cell_num = grid.DoF();
		//real far_from_intf_phi_val=grid.dx*(real)5;

		//// Step 1: find interface cells
#pragma omp parallel for
		for (int i = 0; i < cell_num; i++) {
			const VectorDi cell = grid.Coord(i);
			//if(abs(phi(cell))>far_from_intf_phi_val)continue;		////ATTENTION: this might cause problem if the levelset is badly initialized

			for (int j = 0; j < Grid<d>::Neighbor_Node_Number(); j++) {
				VectorDi nb = grid.Neighbor_Node(cell, j);
				if (!grid.Valid(nb))continue;
				if (Is_Interface(cell, nb)) {
					done[i] = true; break;
				}
			}
		}
		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM Precond: find interface");

		//// Step 2: calculate initial phi values for interface cells
#pragma omp parallel for
		for (int c = 0; c < cell_num; c++) {
			if (!done[c])continue;		////select interface cells
			const VectorDi cell = grid.Coord(c);

			VectorD correct_phi = VectorD::Ones() * std::numeric_limits<real>::max();
			VectorDi correct_axis = VectorDi::Zero();
			for (int i = 0; i < Grid<d>::Neighbor_Node_Number(); i++) {
				VectorDi nb = grid.Neighbor_Node(cell, i);
				if (!grid.Valid(nb)) continue;
				const int nb_idx = grid.Index(nb);
				if (done[nb_idx] && Is_Interface(cell, nb)) {
					real c_phi = Theta(phi(cell), phi(nb)) * grid.dx; // always non-negative
					int axis = grid.Neighbor_Node_Axis(i);
					correct_axis[axis] = 1;
					correct_phi[axis] = std::min(correct_phi[axis], c_phi);
				}
			}
			if (correct_axis != VectorDi::Zero()) {
				real hmnc_mean = (real)0;
				for (int i = 0; i < d; i++) {
					if (correct_axis[i] == 0)continue;
					hmnc_mean += (real)1 / (correct_phi[i] * correct_phi[i]);
				}
				hmnc_mean = sqrt((real)1 / hmnc_mean);
				tent(cell) = hmnc_mean;
			}
			else {
				Error("[Levelset] bad preconditioning");
			}
		}

		//// Step 3: perform relaxation on interface cells to fix their values

		//// initialize heap with front cells
#pragma omp parallel for
		for (int i = 0; i < cell_num; i++) {
			const VectorDi cell = grid.Coord(i);
			if (!done[i]) continue;
			auto [relax_success, val] = Relax_Node(cell, phi, tent, done);
#pragma omp critical
			{heaps[MathFunc::Sign(phi(cell)) > 0 ? 0 : 1].push(PRI(tent(cell), i)); }
		}

		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM: Build heap");

		//// Step 4: relax the other part of field
#pragma omp parallel for
		for (int h = 0; h < 2; h++) {
			auto& heap = heaps[h];
			while (!heap.empty()) {
				const real top_val = heap.top().first;
				const int cell_idx = heap.top().second;
				const VectorDi cell = grid.Coord(cell_idx);
				heap.pop();
				if (tent(cell) != top_val) continue;
				done[cell_idx] = true;

				for (int i = 0; i < Grid<d>::Neighbor_Node_Number(); i++) {
					VectorDi nb = grid.Neighbor_Node(cell, i);
					if (!grid.Valid(nb))continue;
					const int nb_idx = grid.Index(nb);
					//relaxation
					if (!done[nb_idx]) {
						auto [relaxed, val] = Relax_Node(nb, phi, tent, done);
						if (relaxed) heap.push(PRI(val, nb_idx));
					}
				}
			}
		}

		//if (verbose)timer.Elapse_And_Output_And_Reset("FMM: Traverse heap");

		ArrayFunc::Binary_Transform(
			phi.Data(),
			tent.Data(),
			[=](const real phi_i, const real tent_i) {return MathFunc::Sign(phi_i) * tent_i; },
			phi.Data()
		);
	}

	template class LevelSet<2>;
	template class LevelSet<3>;

}