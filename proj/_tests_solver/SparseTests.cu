#include "hip/hip_runtime.h"
#include "SparseTests.h"
#include "ConjugateGradient.h"
#include "Random.h"

void Test_Sparse_Matrix(void)
{
    // test the solve for a sparse matrix
    // rows and cols number of the matrx
    int rows = 10;
    int cols = 10;

    //create a diagonal dominant matrix
    Eigen::SparseMatrix<real, Eigen::RowMajor, int> A;
    Random::Sparse_Diagonal_Dominant_Matrix(rows, cols, A);

    //create b through x to make sure a solution exists
    VectorXd x=Random::Random_VectorXd(cols);
    VectorXd b = A * x;
    //std::cout << "A: \n" << A.toDense() << std::endl;
    //std::cout << "x:  " << x.transpose() << std::endl;
    //std::cout << "b:  " << b.transpose() << std::endl;

    //Solve with Eigen
    Eigen::ConjugateGradient<SparseMatrix<real>, Eigen::Lower | Eigen::Upper, Eigen::IdentityPreconditioner> e_cg;
    e_cg.compute(A);
    x = e_cg.solve(b);
    //std::cout << "Eigen CG solve iterations:     " << e_cg.iterations() << std::endl;
    //std::cout << "Eigen CG solve estimated error: " << e_cg.error() << std::endl;
    //std::cout << "Eigen CG solved x:" << x.transpose() << std::endl;

    //Solve with our CG solver with linear mapping
    SparseMatrixMapping<real, DEVICE> smm(A);
    ConjugateGradient<real> cg;
    cg.verbose = true;
    cg.Init(&smm, nullptr, e_cg.maxIterations(), e_cg.tolerance()); //use the same max iteration and tolerance as Eigen

    //Verify linear mapping first
    ArrayDv<real> x_d(cols), b_d(cols);
    for (int i = 0; i < cols; i++) { x_d[i] = x[i]; }
    smm.Apply(b_d, x_d);

    VectorXd x_cg(cols);
    for (int i = 0; i < cols; i++) { x_cg[i] = x_d[i]; }

    Assert(x_cg.isApprox(x), "Test_Sparse_Matrix: sparse mapped Ap not equal to b");

    ArrayFunc::Copy(b_d, b);

    //Verify our CG solver
    int iters = 0;
    real relative_error = 0;
    cg.Solve(x_d, b_d, iters, relative_error);
    for (int i = 0; i < cols; i++) { x_cg[i] = x_d[i]; }

    if (x_cg.isApprox(x)) {
        Pass("Test_Sparse_Matrix passed");
    }
    else {
        Error("Incorrect Result!");
        std::cout << "Our x:" << x_cg.transpose() << std::endl;
    }
}

void Test_CG_Memory_Safe(void) {
    int rows = 10;
    int cols = 10;

    //create a diagonal dominant matrix
    Eigen::SparseMatrix<real, Eigen::RowMajor, int> A;
    Random::Sparse_Diagonal_Dominant_Matrix(rows, cols, A);

    SparseMatrixMapping<real, DEVICE> smm_A(A);
    ConjugateGradient<real> cg;
    cg.verbose = true;
    cg.Init(&smm_A, nullptr, 1000, 1e-6);

    Eigen::SparseMatrix<real, Eigen::RowMajor, int> B;
    Random::Sparse_Diagonal_Dominant_Matrix(rows, cols, A);
    SparseMatrixMapping<real, DEVICE> smm_B(B);
    cg.Init(&smm_B, nullptr, 1000, 1e-6);
    Pass("Passed initializing memory for multiple times!");
}