#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////
// Nonlinear Thin Shell FEM
// Copyright (c) (2021-), Fan Feng
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////
#include "SoftBodyNonlinearFemThinShell.h"
#include "MeshFunc.h"
#include "AuxFunc.h"
#include "NonlinearFemFunc.h"
#include "Timer.h"
#include "Hashtable.h"
#include "SimplicialPrimitives.h"
#include "IOHelper.h"
#include "Common.h"
#include <Eigen/IterativeLinearSolvers>
#include <iostream>
#include <fstream>

using namespace ThinShellAuxFunc; 
using namespace Meso;

template<class T_ARRAY> int Element_Edges(const Vector2i& v,T_ARRAY& edges);
template<class T_ARRAY> int Element_Edges(const Vector3i& v,T_ARRAY& edges);
void Grad_Q(const ArrayF<Vector3, 3>& vtx, const int i, const int j, const Vector3& ps, const real& qs_i, const ArrayF<Vector3, 3>& ls, const real& a, Vector3& grad_q);
void Grad_R(const ArrayF<Vector3, 3>& vtx, const int i, const int j, const Vector3& ps, const ArrayF<Vector3, 3>& ls, const real& a, ArrayF<Vector3, 2>& grad_r);
void Grad_N(const ArrayF<Vector3, 3>& vtx, const ArrayF<Vector3, 3>& ls, const real& a, ArrayF<Matrix3, 3>& grad_n);

template<int d> real SoftBodyNonlinearFemThinShell<d>::CFL_Time(const real cfl) {
	return 0.01;
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Output(const bf::path base_path, const int frame) {
	std::string vtu_name = fmt::format("vtu{:04d}.vtu", frame);
	bf::path vtu_path = base_path / bf::path(vtu_name);
	VTKFunc::Output_VTU<d, VectorD>(mesh, V(), vtu_path.string());
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Advance(const int current_frame, const real current_time, const real dt) {
	if (use_explicit) { Advance_Explicit(dt); }
	else { Advance_Implicit(dt); }
	return;
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Initialize(SurfaceMesh<d>& _mesh)
{
	mesh=std::make_shared<SurfaceMesh<d>>(particles.XPtr());
	particles.Resize((int)_mesh.Vertices().size());
	*mesh=_mesh;
	MeshFunc::Get_Edges<d,d>(_mesh,edges); //no repetition in edges
	
	//initialize edge hashtable
	for(int i=0;i<Ele_Num();i++){
		VectorDi& vtx_indices=E()[i];
		ArrayF<Vector<int,d-1>,d> edge_set; 
		Element_Edges(vtx_indices,edge_set);
		for(int k=0;k<d;k++){
			Add(edge_element_hashtable,Unique_Ordered(edge_set[k]),i);
		}
	}

	int vtx_n=Vtx_Num();int dof_n=vtx_n*d;int ele_n=Ele_Num();
	material_id.resize(ele_n,0);

	////initialize X0, Dm_inv, N and M
	Set_Rest_Shape(particles.XRef());
	M.resize(dof_n);for(int i=0;i<M.diagonal().size();i++)M.diagonal()[i]=(real)0;
	Dm_inv.resize(ele_n,MatrixD::Zero());
	N.resize(ele_n, VectorD::Zero());
	areas_hat.resize(ele_n);

	ArrayFunc::Fill(F(),VectorD::Zero());
	ArrayFunc::Fill(V(),VectorD::Zero());

	////initialize implicit variables
	if (!use_explicit) {
		A.resize(dof_n,dof_n);
		Allocate_A();
		dv.resize(dof_n);dv.fill((real)0);
		b.resize(dof_n);b.fill((real)0);

		if constexpr (d == 3) {
			if (use_exact_hessian) {
				ls.resize(ele_n);//fill?
				l_norms.resize(ele_n);
				areas.resize(ele_n);
				ps.resize(ele_n, Vector3::Zero());
				hts.resize(ele_n, Vector3::Zero());
				rs.resize(ele_n);//fill?
				grad_ns.resize(ele_n);
				grad_rs.resize(ele_n);
			}
		}
	}
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Allocate_A() {
	std::vector<Triplet<real>> triplets;

	//vertex with itself
	for (int i = 0; i < Vtx_Num(); i++) {
		int r = i; int c = i;
		for (int rr = r * d; rr < (r + 1) * d; rr++)for (int cc = c * d; cc < (c + 1) * d; cc++) {
			triplets.push_back(Triplet<real>(rr, cc, (real)0));
		}
	}

	//vertex with neighboring vertices
	for (int i = 0; i < edges.size(); i++) {
		const Vector2i& e = edges[i]; int r = e[0]; int c = e[1];
		for (int rr = r * d; rr < (r + 1) * d; rr++)for (int cc = c * d; cc < (c + 1) * d; cc++) {
			triplets.push_back(Triplet<real>(rr, cc, (real)0));
		}
		r = e[1]; c = e[0];
		for (int rr = r * d; rr < (r + 1) * d; rr++)for (int cc = c * d; cc < (c + 1) * d; cc++) {
			triplets.push_back(Triplet<real>(rr, cc, (real)0));
		}

		if constexpr (d == 3) {
			Array<int> incident_elements;
			Value_Array(edge_element_hashtable, edges[i], incident_elements);
			if (incident_elements.size() == 2) {
				int face_idx_0 = (incident_elements[0]), face_idx_1 = (incident_elements[1]);
				
				//Find the two vertices at opposite sides
				r = Third_Vertex(e[0], e[1], E()[face_idx_0]);
				c = Third_Vertex(e[0], e[1], E()[face_idx_1]);
				Assert(r != -1 && c != -1, "index {} {} out of range for finding opposite vertex");
				for (int rr = r * d; rr < (r + 1) * d; rr++)for (int cc = c * d; cc < (c + 1) * d; cc++) {
					triplets.push_back(Triplet<real>(rr, cc, (real)0));
					triplets.push_back(Triplet<real>(cc, rr, (real)0));
				}
			}
		}
	}

	A.setFromTriplets(triplets.begin(), triplets.end());
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Initialize_Material() {
	//initialize thickness
	hs.resize(Vtx_Num(), thickness);
	for (int i = 0; i < Ele_Num(); i++) {
		ArrayF<VectorD, d> vtx;
		for (int j = 0; j < d; j++) { vtx[j] = X0[E()[i][j]]; }
		NonlinearFemFunc<d>::D_Inv_And_Area_And_Normal(vtx, Dm_inv[i], areas_hat[i], N[i]);

		for (int j = 0; j < d; j++) {
			int v_idx = E()[i][j];
			for (int k = 0; k < d; k++) {
				M.diagonal()[v_idx * d + k] += hs[v_idx] * areas_hat[i] / (real)(d);	//Mass on vertices
			}
		}
	}

	//initialize lambda, theta
	if constexpr (d == 3) {
		lambdas.resize(edges.size(),(real)1);
		theta_hats.resize(edges.size(),(real)0);

		for(int edge_idx=0; edge_idx <edges.size(); edge_idx++){
			ArrayF<int,d+1> vtx_idx;
			ArrayF<int, 2> ele_idx;
			if (Junction_Info(edge_idx, vtx_idx, ele_idx)) { //shared edge
				real l_hat = (X()[vtx_idx[0]] - X()[vtx_idx[1]]).norm();
				real a_hat = areas_hat[ele_idx[0]]+ areas_hat[ele_idx[1]];
				ElasticParam& material0=materials[material_id[ele_idx[0]]];
				ElasticParam& material1=materials[material_id[ele_idx[1]]];
				real avg_h0, avg_h1, avg_h;
				avg_h0 = avg_h1 = avg_h = 0;
				for (int i = 0; i < 4; i++) { avg_h += hs[vtx_idx[i]]; } avg_h /= (real)4;
				for (int i = 0; i < 3; i++) { avg_h0 += hs[E()[ele_idx[0]][i]]; } avg_h0 /= (real)3;
				for (int i = 0; i < 3; i++) { avg_h1 += hs[E()[ele_idx[1]][i]]; } avg_h1 /= (real)3;
				real ks0=Ks(material0.youngs_modulus, avg_h,material0.poisson_ratio);	//this thickness should use the thickness on face instead?
				real ks1=Ks(material1.youngs_modulus, avg_h,material1.poisson_ratio);	//although it really simplifies the calculation
				lambdas[edge_idx]=Lambda(Kb(avg_h,(real)0.5*(ks0+ks1)),a_hat,l_hat);
				VectorD n0=Triangle<3>::Normal(X()[E()[ele_idx[0]][0]],X()[E()[ele_idx[0]][1]],X()[E()[ele_idx[0]][2]]);
				VectorD n1=Triangle<3>::Normal(X()[E()[ele_idx[1]][0]],X()[E()[ele_idx[1]][1]],X()[E()[ele_idx[1]][2]]);
				theta_hats[edge_idx]=Dihedral_Angle(n0,n1,X()[vtx_idx[0]],X()[vtx_idx[1]]);
			}
		}
	}
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Add_Material(real youngs,real poisson)
{materials.push_back(ElasticParam(youngs,poisson));}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Set_Fixed(const int node)
{bc.psi_D_values[node]=VectorD::Zero();}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Set_Displacement(const int node,const VectorD& dis)
{bc.psi_D_values[node]=dis;}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Set_Force(const int node,const VectorD& force)
{bc.forces[node]=force;}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Add_Force(const int node,const VectorD& force)
{bc.forces[node]+=force;}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Clear_Force()
{bc.forces.clear();}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Set_Rest_Shape(const Array<VectorD>& _X0)
{X0=_X0;}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Advance_Explicit(const real dt)
{
	Timer timer;

	ArrayFunc::Fill(F(),VectorD::Zero());
	const int vtx_num=Vtx_Num();

	////body force, damping force, boundary
	if(use_body_force){for(int i=0;i<vtx_num;i++)F()[i]+=Mass(i)*g; }

	//damping
	for(int i=0;i<vtx_num;i++)F()[i]-=Mass(i)*damping*V()[i];

	//stretching forces
	const int ele_num = Ele_Num();
	for(int ele_idx=0; ele_idx < ele_num; ele_idx++){
		MatrixD grad_s;
		Grad_Stretch(ele_idx,grad_s);
		for(int j=0;j<d;j++){F()[E()[ele_idx][j]]-=grad_s.col(j);}

		/*real stretching_energy=Stretching_Energy<d>(ele_idx);
		MatrixD grad_s_n=Numerical_Grad_Stretch(ele_idx,stretching_energy,grad_s);*/
	}

	////bending forces, only support for three dimension for now
	if constexpr (d == 3) {
		for(int i=0;i<edges.size();i++){
			Eigen::Matrix<real,d,d+1> grad_b;
			ArrayF<int,d+1> vtx_idx;
			ArrayF<int, 2> ele_idx;
			if (Junction_Info(i, vtx_idx, ele_idx)) {
				Grad_Bend(i, grad_b, vtx_idx, ele_idx);
				for(int j=0;j<d+1;j++){
					F()[vtx_idx[j]]-=grad_b.col(j);
				}

				/*Eigen::Matrix<real,d,d+1> grad_b_n=Numerical_Grad_Bend(i,theta_hats[i],lambdas[i]);
				std::cout<<"grad_b["<<i<<"]: \n"<<grad_b<<"\n"<<"grad_b_n["<<i<<"]: \n"<<grad_b_n<<std::endl;*/
			}
		}
	}

	//enforce boundary condition
	for(auto& iter:bc.forces){F()[iter.first]+=iter.second;}

	////time integration
	for(int i=0;i<vtx_num;i++){
		V()[i]+=F()[i]/Mass(i)*dt;
		if(bc.Is_Psi_D(i))V()[i]=VectorD::Zero();
		X()[i]+=V()[i]*dt;}
	Info("Explicit time integration: {} ms" , timer.Total_Time(PhysicalUnits::ms));
}

// A = dt^2 J + dt damp J
// b = dt f + dt^2 J v + dt damp J v
template<int d> void SoftBodyNonlinearFemThinShell<d>::Advance_Implicit(const real dt)
{
	Timer timer;

	Clear_A_And_Rhs();
	std::cout << "# number of non-zeros of A:  " << A.nonZeros() << std::endl;
	const int vtx_num=Vtx_Num();
	const int ele_num=Ele_Num();

	Update_Implicit_Force_And_Mass(dt);
	Update_Implicit_Stretching(dt);
	Update_Implicit_Bending(dt);
	Update_Implicit_Boundary_Condition(dt);

	Eigen::ConjugateGradient<SparseMatrix<real>, Eigen::Lower | Eigen::Upper, Eigen::DiagonalPreconditioner<real>> cg;
	cg.setTolerance((real)1e-5);
	dv = cg.compute(A).solve(b);
	Info("Linear system solve: {} ms", timer.Lap_Time(PhysicalUnits::ms));

	std::cout << "#	CG iterations:     " << cg.iterations() << std::endl;
	std::cout << "#	CG estimated error: " << cg.error() << std::endl;

	//Eigen::LLT<Eigen::MatrixXd> lltOfA(A); // compute the Cholesky decomposition of A
	//if (lltOfA.info() == Eigen::NumericalIssue){throw std::runtime_error("Negative matrix!");}
	//if (!A.isApprox(A.transpose())) {throw std::runtime_error("Non symmetric matrix!");}

#pragma omp parallel for
	for (int i = 0; i < vtx_num; i++) {
		for (int j = 0; j < d; j++) { V()[i][j] += dv[i * d + j]; }
		X()[i] += V()[i] * dt;
	}

	Info("update nodes: {} ms", timer.Lap_Time(PhysicalUnits::ms));
}

// A = hess
// b = -grad
// Adx=b
template<int d> void SoftBodyNonlinearFemThinShell<d>::Advance_Quasi_Static()
{	//dv is dx in this case
	int iter = 0;
	real err = 1;
	const int vtx_num = Vtx_Num();
	const int ele_num = Ele_Num();
	const real alpha = 0.1;
	real energy = 0;
	const int max_iter = 1000;
	while (err > 1e-4) {
		if (iter == max_iter) {
			Info("max iteration {} reached!",max_iter);
			break;
		}
		//energy = (real)0;
		//Info("");
		//Info("Start the {}th Newton iteration: ", iter);
		//Timer timer;
		//timer.Reset();

		SparseFunc::Set_Value(A, (real)0);
		b.setZero(); //dense vector b can be directly set to zero
		//std::cout << "# number of non-zeros of A:  " << A.nonZeros() << std::endl;

		//add external forces
		for (auto& force : bc.forces) {
			Add_Block(b, force.first, force.second);
			//energy += -force.second.dot(X()[force.first]); //potential energy by the external force
		}
		//timer.Elapse_And_Output_And_Reset("Add external force for b");

		//Stretching
		//Timer timer2;
		//timer2.Begin_Loop();
		for (int ele_idx = 0; ele_idx < ele_num; ele_idx++) {
			MatrixD grad_s; MatrixD hess_s[d][d];
			Stretch_Force(ele_idx, grad_s, hess_s);

			/*MatrixD grad_s_n = Numerical_Grad_Stretch(ele_idx,grad_s);
			Array2DF<Matrix<real, d>, d, d> hess_s_n = Numerical_Hess_Stretch(ele_idx, grad_s,hess_s);*/

			//timer2.Record("calculate streching force");

			//iterate through verteices in the element
			for (int j = 0; j < d; j++) {
				Add_Block(b, E()[ele_idx][j], -grad_s.col(j));
				for (int k = 0; k < d; k++) {
					Add_Block_Helper(A, E()[ele_idx][j], E()[ele_idx][k], hess_s[j][k]);
				}
			}
			//timer2.Record("assemble strething matrix");

			//energy += Stretching_Energy(areas_hat[ele_idx], ks, material.poisson_ratio,strain);
		}

		//timer2.End_Loop_And_Output(std::cout);
		//timer.Elapse_And_Output_And_Reset("Assemble linear system for stretching");

		//Bending
		//timer2.Begin_Loop();
		if constexpr (d == 3) {
			Update_Bending_Hess_Variables();
			for (int jct_idx = 0; jct_idx < edges.size(); jct_idx++) {
				Eigen::Matrix<real, d, d + 1> grad_b;
				MatrixD hess_b[d + 1][d + 1];
				ArrayF<int, d + 1> vtx_idx;
				ArrayF<int, 2> ele_idx;
				if (Junction_Info(jct_idx, vtx_idx, ele_idx)) {
					Bend_Force(jct_idx, grad_b, vtx_idx, ele_idx, hess_b);
					//timer2.Record("calculate bending force");
					for (int j = 0; j < d + 1; j++) {
						Add_Block(b, vtx_idx[j], -grad_b.col(j));
						for (int k = 0; k < d + 1; k++) {
							Add_Block_Helper(A, vtx_idx[j], vtx_idx[k], hess_b[j][k]);
						}
					}

					/*Eigen::Matrix<real,d,d+1> grad_b_n=Numerical_Grad_Bend(jct_idx,vtx_idx,ele_idx,grad_b);
					Array2DF<Matrix<real, d>, d + 1, d + 1> hess_b_n = Numerical_Hess_Bend(jct_idx, vtx_idx, ele_idx, grad_b,hess_b);*/

					//timer2.Record("assemble bending matrix");

					//extra calculation for bending energy, could be simplified with the bend force calculation?
					/*Vector3 n0 = Triangle<3>::Normal(X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]]);
					Vector3 n1 = Triangle<3>::Normal(X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]]);
					real theta = Dihedral_Angle(n0, n1, X()[vtx_idx[0]], X()[vtx_idx[1]]);

					energy += Bending_Energy(lambdas[jct_idx], theta, theta_hats[jct_idx]);*/
				}
			}
		}
		//timer2.End_Loop_And_Output(std::cout);
		//timer.Elapse_And_Output_And_Reset("Assemble linear system for bending");
		
		for (auto& bc_d : bc.psi_D_values) {
			int node = bc_d.first; VectorD dis = bc_d.second;
			for (int axis = 0; axis < d; axis++) {
				int idx = node * d + axis;
				if(iter==0){ NonlinearFemFunc<d>::Set_Dirichlet_Boundary_Helper(A, b, idx, dis[axis]); }
				else{ NonlinearFemFunc<d>::Set_Dirichlet_Boundary_Helper(A, b, idx, (real)0); }
			}
		}
		
		Eigen::ConjugateGradient<SparseMatrix<real>, Eigen::Lower | Eigen::Upper> cg;
		cg.setTolerance((real)1e-6);
		cg.compute(A);
		dv = cg.solve(b);

		///*std::cout << "A:" << std::endl;
		//std::cout << A << std::endl;
		//std::cout << "b:" << std::endl;
		//std::cout << b.transpose() << std::endl;
		//std::cout << "dv:" << std::endl;
		//std::cout << dv.transpose() << std::endl;*/
		//
		////timer.Elapse_And_Output_And_Reset("linear system solve");

		////std::cout << "#	CG iterations:     " << cg.iterations() << std::endl;
		////std::cout << "#	CG estimated error: " << cg.error() << std::endl;

		//#pragma omp parallel for
		//for (int i = 0; i < vtx_num; i++) {
		//	for (int j = 0; j < d; j++) {
		//		X()[i][j] += damping*dv[i * d + j]; //damping becomes the step size here
		//	}
		//}

		////timer.Elapse_And_Output_And_Reset("update nodes");

		//err = dv.norm() / dv.size();
		////Info("b norm is:{}", b.norm());
		////Info("relative error is:{}", err);
		//iter++;
		////energies_n.push_back(energy);
	}
	Info("Quasi_static solve finished with {} Newton iterations: ", iter);
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Clear_A_And_Rhs(){
	SparseFunc::Set_Value(A, (real)0);
	b.setZero();
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Update_Implicit_Force_And_Mass(const real dt){
	Timer timer;
	const int vtx_num=Vtx_Num();
	const int ele_num=Ele_Num();

	//add external forces
	if(use_body_force){
		for (int i = 0; i < vtx_num; i++) {
			Add_Block(b, i, dt * Mass(i) * g);
		}
	}
	Info("Add body force for b", timer.Lap_Time(PhysicalUnits::ms));

	for(auto& iter:bc.forces){
		Add_Block(b,iter.first, dt*iter.second);
	}
	Info("Add external force for b", timer.Lap_Time(PhysicalUnits::ms));

	for(int i=0;i<vtx_num;i++){Add_Block_Helper(A,i,i,Mass(i)*MatrixD::Identity());}
	Info("Add mass for A", timer.Lap_Time(PhysicalUnits::ms));
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Update_Implicit_Stretching(const real dt){
	Timer timer;
	const int vtx_num=Vtx_Num();
	const int ele_num=Ele_Num();
		
	Timer timer2;
	timer2.Begin_Loop();
	for(int ele_idx=0; ele_idx <ele_num; ele_idx++){
		MatrixD grad_s; MatrixD hess_s[d][d];
		Stretch_Force(ele_idx, grad_s, hess_s);
		timer2.Record("calculate streching force");
		for(int j=0;j<d;j++){
			Add_Block(b, E()[ele_idx][j], -dt*grad_s.col(j));
			for(int k=0;k<d;k++){
				Add_Block(b,E()[ele_idx][j],-dt*(dt+damping)*hess_s[j][k]*V()[E()[ele_idx][k]]);
				Add_Block_Helper(A, E()[ele_idx][j], E()[ele_idx][k], dt * (dt + damping) * hess_s[j][k]);
			}
		}
		timer2.Record("assemble strething matrix");
	}
	timer2.Output_Profile(std::cout);
	Info("Assemble linear system for stretching", timer.Lap_Time(PhysicalUnits::ms));
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Update_Implicit_Bending(const real dt) {
	Timer timer;
	const int vtx_num = Vtx_Num();
	const int ele_num = Ele_Num();

	timer.Begin_Loop();
	if constexpr (d == 3) {
		for (int i = 0; i < edges.size(); i++) {
			Eigen::Matrix<real, d, d + 1> grad_b;
			Eigen::Matrix<real, d, d> hess_b[d + 1][d + 1];
			ArrayF<int, d + 1> vtx_idx;
			ArrayF<int, 2> ele_idx;
			if (Junction_Info(i, vtx_idx, ele_idx)) {
				Bend_Force_Approx(i, grad_b, vtx_idx, ele_idx, hess_b);
				timer.Record("calculate bending force");
				for (int j = 0; j < d + 1; j++) {
					Add_Block(b, vtx_idx[j], -dt * grad_b.col(j));

					for (int k = 0; k < d + 1; k++) {
						Add_Block(b, vtx_idx[j], -dt * (dt + damping) * hess_b[j][k] * V()[vtx_idx[k]]);
						Add_Block_Helper(A, vtx_idx[j], vtx_idx[k], dt * (dt + damping) * hess_b[j][k]);
					}
				}
				timer.Record("assemble bending matrix");
			}
		}
	}
	timer.Output_Profile(std::cout);
	Info("Assemble linear system for bending", timer.Lap_Time(PhysicalUnits::ms));
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Update_Implicit_Boundary_Condition(const real dt){
	Timer timer;
	const int vtx_num=Vtx_Num();
	const int ele_num=Ele_Num();

	for (auto& bc_d : bc.psi_D_values) {
		int node = bc_d.first; VectorD dis = bc_d.second;
		for (int axis = 0; axis < d; axis++) {
			int idx = node * d + axis;
			NonlinearFemFunc<d>::Set_Dirichlet_Boundary_Helper(A, b, idx, dis[axis]);
		}
	}
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Grad_Stretch(real area, const MatrixD& stress, const MatrixD& x_hat, const MatrixD& Dm_inv, MatrixD& grad) {
	MatrixD P=area*stress;
	MatrixD C_x=C_c()*Dm_inv;
	grad=x_hat*C_x*P*C_x.transpose();
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Grad_Stretch(const int ele_idx, MatrixD& grad_s) {
	ElasticParam& material = materials[material_id[ele_idx]];
	real avg_h = 0; //average thickness on an element, not considering weighing yet

	ArrayF<VectorD, d> vtx;
	MatrixD x_hat;
	for (int j = 0; j < d; j++) {
		vtx[j] = X()[E()[ele_idx][j]];
		x_hat.col(j) = vtx[j];
		avg_h += hs[E()[ele_idx][j]];
	}

	avg_h *= (real)1 / (real)d;
	real ks = Ks(material.youngs_modulus, avg_h, material.poisson_ratio);

	MatrixD ds;
	NonlinearFemFunc<d>::D(vtx, ds);
	MatrixD deformation = ds * Dm_inv[ele_idx];
	MatrixD strain;
	Deformation_To_Strain(deformation, strain);
	MatrixD stress;
	Strain_To_Stress(ks, material.poisson_ratio, strain, stress);

	Grad_Stretch(areas_hat[ele_idx], stress, x_hat, Dm_inv[ele_idx], grad_s);
}


template<int d> void SoftBodyNonlinearFemThinShell<d>::Stretch_Force(const real& area,const real& ks, const real& poisson_ratio, const MatrixD& stress, const MatrixD& x_hat, const MatrixD& Dm_inv, MatrixD& grad, MatrixD hess[d][d]) {
	MatrixD P = area * stress;
	MatrixD C_x = C_c() * Dm_inv;
	grad = x_hat * C_x * P * C_x.transpose();

	MatrixD R = C_x * P * C_x.transpose();
	MatrixD Q = C_x * C_x.transpose();
	MatrixD y_hat = x_hat * Q;
	MatrixD z_hat = y_hat * x_hat.transpose();

	for (int i = 0; i < d; i++) {
		for (int j = 0; j < d; j++) {
			for (int p = 0; p < d; p++) {
				for (int q = 0; q < d; q++) {
					hess[j][q](i, p) = ((i == p) ? R(q, j) : (real)0) + area * ks * poisson_ratio * y_hat(i, j) * y_hat(p, q) + (real)0.5 * area * ks * (1 - poisson_ratio) * (z_hat(p, i) * Q(q, j) + y_hat(p, j) * y_hat(i, q));
				}
			}
		}
	}
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Stretch_Force(const int ele_idx, MatrixD& grad_s, MatrixD hess_s[d][d]) {
	ElasticParam& material = materials[material_id[ele_idx]];
	real avg_h = 0; //average thickness on an element, not considering weighing yet

	ArrayF<VectorD, d> vtx;
	MatrixD x_hat;
	for (int j = 0; j < d; j++) {
		vtx[j] = X()[E()[ele_idx][j]];
		x_hat.col(j) = vtx[j];
		avg_h += hs[E()[ele_idx][j]];
	}

	avg_h *= (real)1 / (real)d;
	real ks = Ks(material.youngs_modulus, avg_h, material.poisson_ratio);

	MatrixD ds;
	NonlinearFemFunc<d>::D(vtx, ds);
	MatrixD deformation = ds * Dm_inv[ele_idx];
	MatrixD strain;
	Deformation_To_Strain(deformation, strain);
	MatrixD stress;
	Strain_To_Stress(ks, material.poisson_ratio, strain, stress);

	Stretch_Force(areas_hat[ele_idx], ks, material.poisson_ratio, stress, x_hat, Dm_inv[ele_idx], grad_s, hess_s);
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Grad_Bend(const Eigen::Matrix<real,d,d+1>& dtheta_dx, real theta, real theta_hat, real lambda, Eigen::Matrix<real,d,d+1>& dE_dx){
	dE_dx=(real)2*lambda*(theta-theta_hat)*dtheta_dx;
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Grad_Bend(int jct_idx, Eigen::Matrix<real,d,d+1>& grad, const ArrayF<int,d+1>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	//no implementation
}

template<> void SoftBodyNonlinearFemThinShell<2>::Grad_Bend(int jct_idx, Eigen::Matrix<real,2,3>& grad, const ArrayF<int,3>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	return; //not implemented yet
}

template<> void SoftBodyNonlinearFemThinShell<3>::Grad_Bend(int jct_idx, Eigen::Matrix<real,3,4>& grad, const ArrayF<int,4>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	Vector3 n0=Triangle<3>::Normal(X()[E()[ele_idx[0]][0]],X()[E()[ele_idx[0]][1]],X()[E()[ele_idx[0]][2]]);
	Vector3 n1=Triangle<3>::Normal(X()[E()[ele_idx[1]][0]],X()[E()[ele_idx[1]][1]],X()[E()[ele_idx[1]][2]]);

	real theta= Dihedral_Angle(n0,n1,X()[vtx_idx[0]],X()[vtx_idx[1]]);

	Vector2 w0=Barycentric_Weights(X()[vtx_idx[2]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	Vector2 w1=Barycentric_Weights(X()[vtx_idx[3]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	real h0=Distance(X()[vtx_idx[2]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	real h1=Distance(X()[vtx_idx[3]],X()[vtx_idx[0]],X()[vtx_idx[1]]);

	Eigen::Matrix<real,3,4> dtheta;
	dtheta.col(0)=-(w0[0]*n0/h0+w1[0]*n1/h1);
	dtheta.col(1)=-(w0[1]*n0/h0+w1[1]*n1/h1);
	dtheta.col(2)=n0/h0;
	dtheta.col(3)=n1/h1;
	grad=(real)2*lambdas[jct_idx]*(theta-theta_hats[jct_idx])*dtheta;
}

template<> void SoftBodyNonlinearFemThinShell<2>::Grad_Theta(Eigen::Matrix<real, 2, 3>& grad_theta, const ArrayF<int, 3>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	return; //no implementation
}

template<> void SoftBodyNonlinearFemThinShell<3>::Grad_Theta(Eigen::Matrix<real, 3, 4>& grad_theta, const ArrayF<int, 4>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	Vector3 n0 = Triangle<3>::Normal(X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]]);
	Vector3 n1 = Triangle<3>::Normal(X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]]);

	real theta = Dihedral_Angle(n0, n1, X()[vtx_idx[0]], X()[vtx_idx[1]]);

	Vector2 w0 = Barycentric_Weights(X()[vtx_idx[2]], X()[vtx_idx[0]], X()[vtx_idx[1]]);
	Vector2 w1 = Barycentric_Weights(X()[vtx_idx[3]], X()[vtx_idx[0]], X()[vtx_idx[1]]);
	real h0 = Distance(X()[vtx_idx[2]], X()[vtx_idx[0]], X()[vtx_idx[1]]);
	real h1 = Distance(X()[vtx_idx[3]], X()[vtx_idx[0]], X()[vtx_idx[1]]);

	grad_theta.col(0) = -(w0[0] * n0 / h0 + w1[0] * n1 / h1);
	grad_theta.col(1) = -(w0[1] * n0 / h0 + w1[1] * n1 / h1);
	grad_theta.col(2) = n0 / h0;
	grad_theta.col(3) = n1 / h1;
}

template<> void SoftBodyNonlinearFemThinShell<2>::Bend_Force_Approx(int edge_idx, Eigen::Matrix<real,2,3>& grad, const ArrayF<int,3>& vtx_idx, const ArrayF<int, 2>& ele_idx, MatrixD hess[3][3]) {
	return; //not implemented yet
}

template<> void SoftBodyNonlinearFemThinShell<3>::Bend_Force_Approx(int edge_idx, Eigen::Matrix<real,3,4>& grad, const ArrayF<int,4>& vtx_idx, const ArrayF<int, 2>& ele_idx, MatrixD hess[4][4]) {
	Vector3 n0=Triangle<3>::Normal(X()[E()[ele_idx[0]][0]],X()[E()[ele_idx[0]][1]],X()[E()[ele_idx[0]][2]]);
	Vector3 n1=Triangle<3>::Normal(X()[E()[ele_idx[1]][0]],X()[E()[ele_idx[1]][1]],X()[E()[ele_idx[1]][2]]);

	real theta=Dihedral_Angle(n0,n1,X()[vtx_idx[0]],X()[vtx_idx[1]]);

	Vector2 w0=Barycentric_Weights(X()[vtx_idx[2]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	Vector2 w1=Barycentric_Weights(X()[vtx_idx[3]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	real h0=Distance(X()[vtx_idx[2]],X()[vtx_idx[0]],X()[vtx_idx[1]]);
	real h1=Distance(X()[vtx_idx[3]],X()[vtx_idx[0]],X()[vtx_idx[1]]);

	Eigen::Matrix<real,3,4> dtheta;
	dtheta.col(0)=-(w0[0]*n0/h0+w1[0]*n1/h1);
	dtheta.col(1)=-(w0[1]*n0/h0+w1[1]*n1/h1);
	dtheta.col(2)=n0/h0;
	dtheta.col(3)=n1/h1;

	grad=(real)2*lambdas[edge_idx]*(theta-theta_hats[edge_idx])*dtheta;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			hess[i][j]=(real)2*lambdas[edge_idx]*(dtheta.col(i))*(dtheta.col(j).transpose());
		}
	}
}

template<> void SoftBodyNonlinearFemThinShell<2>::Bend_Force(int jct_idx, Eigen::Matrix<real, 2, 3>& grad, const ArrayF<int, 3>& vtx_idx, const ArrayF<int, 2>& ele_idx, MatrixD hess[3][3]) {
	return; //not implemented yet
}

template<> void SoftBodyNonlinearFemThinShell<3>::Bend_Force(int jct_idx, Eigen::Matrix<real, 3, 4>& grad, const ArrayF<int, 4>& vtx_idx, const ArrayF<int, 2>& ele_idx, MatrixD hess[4][4]) {
	Vector3 n0 = Triangle<3>::Normal(X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]]);
	Vector3 n1 = Triangle<3>::Normal(X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]]);

	real theta = Dihedral_Angle(n0, n1, X()[vtx_idx[0]], X()[vtx_idx[1]]);

	ArrayF<int, 4> map0, map1; //vtx_idx to triangle index
	map0.fill((int)-1); map1.fill((int)-1);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 4; j++) {
			if (E()[ele_idx[0]][i] == vtx_idx[j]) {
				map0[j] = i;
			}

			if (E()[ele_idx[1]][i] == vtx_idx[j]) {
				map1[j] = i;
			}
		}
	}

	real ht0 = hts[ele_idx[0]][map0[2]];
	real ht1 = hts[ele_idx[1]][map1[3]];

	Eigen::Matrix<real, 3, 4> grad_theta;
	const Vector2& r0 = rs[ele_idx[0]][map0[2]];
	const Vector2& r1 = rs[ele_idx[1]][map1[3]];
	grad_theta.col(0) = -(r0[0]*n0  + r1[1]*n1); // reverse order for the second triangle because the shared points should be iterated reversely
	grad_theta.col(1) = -(r0[1]*n0  + r1[0]*n1);
	grad_theta.col(2) = n0 / ht0;
	grad_theta.col(3) = n1 / ht1;

	grad = (real)2 * lambdas[jct_idx] * (theta - theta_hats[jct_idx]) * grad_theta;

	Matrix3 hess_theta[4][4];
	hess_theta[2][3] = hess_theta[3][2] = Matrix3::Zero();

	ArrayF<Vector3, 3>vtx0 {X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]] };
	ArrayF<Vector3, 3>vtx1{ X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]] };
	real q0 = (real)1 / hts[ele_idx[0]][map0[2]];
	real q1 = (real)1 / hts[ele_idx[1]][map1[3]];

	ArrayF<Matrix3, 3>& grad_n0 = grad_ns[ele_idx[0]];
	ArrayF<Matrix3, 3>& grad_n1 = grad_ns[ele_idx[1]];
	
	int i0_p = map0[2]; //mapped i in triangle index
	for (int j = 0; j <= 2; j++) {
		int j_p = map0[j]; //mapped j
		Vector3 grad_q0;
		Grad_Q(vtx0, i0_p, j_p, ps[ele_idx[0]], q0, ls[ele_idx[0]], areas[ele_idx[0]], grad_q0);
		hess_theta[j][2] = grad_q0 * n0.transpose() + q0 * grad_n0[j_p];
		if (2 != j) {hess_theta[2][j] = hess_theta[j][2].transpose();}
	}

	int i1_p = map1[3];
	for (int j = 0; j <= 3; j++) {
		if (j == 2) { continue; }
		int j_p = map1[j];
		Vector3 grad_q1;
		Grad_Q(vtx1, i1_p, j_p, ps[ele_idx[1]], q1, ls[ele_idx[1]], areas[ele_idx[1]], grad_q1);
		hess_theta[j][3] = grad_q1 * n1.transpose() + q1 * grad_n1[j_p];
		if (3 != j) {hess_theta[3][j] = hess_theta[j][3].transpose();}
	}

	
	ArrayF<ArrayF<Vector3, 2>, 3>& grad_r0 = grad_rs[ele_idx[0]][i0_p];
	ArrayF<ArrayF<Vector3, 2>, 3>& grad_r1 = grad_rs[ele_idx[1]][i1_p];

	hess_theta[0][0] = -grad_r0[map0[0]][0] * n0.transpose() - r0[0] * grad_n0[map0[0]] - grad_r1[map1[0]][1] * n1.transpose() - r1[1] * grad_n1[map1[0]];
	hess_theta[1][1] = -grad_r0[map0[1]][1] * n0.transpose() - r0[1] * grad_n0[map0[1]] - grad_r1[map1[1]][0] * n1.transpose() - r1[0] * grad_n1[map1[1]];
	
	hess_theta[1][0] = -grad_r0[map0[1]][0] * n0.transpose() - r0[0] * grad_n0[map0[1]] - grad_r1[map1[1]][1] * n1.transpose() - r1[1] * grad_n1[map1[1]];
	hess_theta[0][1] = hess_theta[1][0].transpose();

	/*std::cout << "hess_theta[" << jct_idx << "]" << std::endl;
	Array2DF<Matrix3, 4, 4> hess_theta_n = Numerical_Hess_Theta(vtx_idx, ele_idx, grad_theta);
	for (int i = 0; i < 4;i++) {
		for (int j = 0; j < 4;j++) {
			if (!hess_theta[i][j].isApprox(hess_theta_n[i][j], (real)1e-2)) {
				std::cout << "hess_theta[" << i << "][" << j << "] \n" << hess_theta[i][j] << "\n" << "hess_theta_n[" << i << "][" << j << "] \n" << hess_theta_n[i][j] << std::endl;
			}
		}
	}*/

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			hess[i][j] = (real)2 * lambdas[jct_idx] * ((theta - theta_hats[jct_idx]) * hess_theta[i][j] + (grad_theta.col(i)) * (grad_theta.col(j).transpose()));
		}
	}
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Update_Bending_Hess_Variables() {
	if constexpr (d == 3) {
#pragma omp parallel for
		for (int ele_idx = 0; ele_idx < Ele_Num(); ele_idx++) {
			ArrayF<VectorD, 3> vtx;
			for (int i = 0; i < 3; i++) { vtx[i] = X()[E()[ele_idx][i]]; }
			for (int i = 0; i < 3; i++) { ls[ele_idx][i] = vtx[(i + 1) % 3] - vtx[(i + 2) % 3]; }
			for (int i = 0; i < 3; i++) { l_norms[ele_idx][i] = ls[ele_idx][i].norm(); }
			for (int i = 0; i < 3; i++) { areas[ele_idx] = Triangle<3>::Area(vtx[0], vtx[1], vtx[2]); }

			for (int i = 0; i < 3; i++) {
				ps[ele_idx][i] = ls[ele_idx][(i + 1) % 3].dot(ls[ele_idx][(i + 1) % 3]) + ls[ele_idx][(i + 2) % 3].dot(ls[ele_idx][(i + 2) % 3]) - ls[ele_idx][i].dot(ls[ele_idx][i]);
			}

			for (int i = 0; i < 3; i++) { hts[ele_idx][i] = (real)2 * areas[ele_idx] / l_norms[ele_idx][i]; }

			for (int i = 0; i < 3; i++) {
				rs[ele_idx][i][0] = ps[ele_idx][(i + 2) % 3] / (real)4 / areas[ele_idx] / l_norms[ele_idx][i];
				rs[ele_idx][i][1] = ps[ele_idx][(i + 1) % 3] / (real)4 / areas[ele_idx] / l_norms[ele_idx][i];
				//method2
				/*Vector2 w = Barycentric_Weights(vtx[i], vtx[(i + 1) % 3], vtx[(i + 2) % 3]);
				rs[ele_idx][i][0] = w[0] / hts[ele_idx][i];
				rs[ele_idx][i][1] = w[1] / hts[ele_idx][i];*/
			}

			Grad_N(vtx, ls[ele_idx], areas[ele_idx], grad_ns[ele_idx]);
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					Grad_R(vtx, i,j,ps[ele_idx], ls[ele_idx], areas[ele_idx], grad_rs[ele_idx][i][j]);
				}
			}
		}
	}
}


template<int d> inline void SoftBodyNonlinearFemThinShell<d>::Reset_To_Rest_Position() {
#pragma omp parallel for
	for (int i = 0; i < Vtx_Num(); i++) {X()[i] = X0[i];}
}

//Helper functions
template<int d> inline void SoftBodyNonlinearFemThinShell<d>::Strain_To_Stress(real ks, real poisson_ratio, const MatrixD& strain,MatrixD& stress)
{
	stress=ks*((1-poisson_ratio)*strain+poisson_ratio*strain.trace()*MatrixD::Identity());
}

template<int d> inline void SoftBodyNonlinearFemThinShell<d>::Deformation_To_Strain(const MatrixD& deformation, MatrixD& strain) {
	strain=(deformation.transpose()*deformation - MatrixD::Identity()) * (real)0.5;
}

template<int d> const Matrix<real,d> SoftBodyNonlinearFemThinShell<d>::C_c() const{/*not impl*/return MatrixD::Identity();}

template<> const Matrix2 SoftBodyNonlinearFemThinShell<2>::C_c() const{
	Matrix2 C_c;
	C_c<<Vector2((real)-1,(real)1),Vector2::Zero();
	return C_c;
}

template<> const Matrix3 SoftBodyNonlinearFemThinShell<3>::C_c() const{
	Matrix3 C_c;
	C_c<<Vector3((real)-1,(real)1,(real)0),Vector3((real)-1,(real)0,(real)1),Vector3::Zero();
	return C_c;
}

template<int d> inline real SoftBodyNonlinearFemThinShell<d>::Stretching_Energy(real a, real ks, real nu, const Matrix<real,d>& strain) {
	real c= (strain.transpose() * strain).trace();
	real b = pow(strain.trace(), 2);
	return (real)0.5*a*ks*(((real)1-nu)*(strain.transpose()*strain).trace() + nu*pow(strain.trace(),2));
}

template<int d> inline real SoftBodyNonlinearFemThinShell<d>::Stretching_Energy(int ele_idx) {
	ElasticParam& material = materials[material_id[ele_idx]];
	real avg_h = 0; //average thickness on an element, not considering weighing yet

	ArrayF<VectorD, d> vtx;
	MatrixD x_hat;
	for (int j = 0; j < d; j++) {
		vtx[j] = X()[E()[ele_idx][j]];
		x_hat.col(j) = vtx[j];
		avg_h += hs[E()[ele_idx][j]];
	}

	avg_h *= (real)1 / (real)d;
	real ks = Ks(material.youngs_modulus, avg_h, material.poisson_ratio);

	MatrixD ds;
	NonlinearFemFunc<d>::D(vtx, ds);
	MatrixD deformation = ds * Dm_inv[ele_idx];
	MatrixD strain;
	Deformation_To_Strain(deformation, strain);

	return Stretching_Energy(areas[ele_idx],ks, material.poisson_ratio,strain);
}

template<int d> inline real SoftBodyNonlinearFemThinShell<d>::Bending_Energy(real lambda, real theta, real theta_hat) {
	return lambda*(theta-theta_hat)*(theta-theta_hat);
}

template<int d> inline real SoftBodyNonlinearFemThinShell<d>::Bending_Energy(int jct_idx, const ArrayF<int, d + 1>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	if constexpr (d == 3) {
		Vector3 n0 = Triangle<3>::Normal(X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]]);
		Vector3 n1 = Triangle<3>::Normal(X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]]);
		real theta = ThinShellAuxFunc::Dihedral_Angle(n0, n1, X()[vtx_idx[0]], X()[vtx_idx[1]]);
		return Bending_Energy(lambdas[jct_idx], theta, theta_hats[jct_idx]);
	}
	else {
		return 0; //to be implemented
	}
}

template<int d> real SoftBodyNonlinearFemThinShell<d>::Total_Stretching_Energy() {
	real stretching_energy = 0;
#pragma omp parallel for reduction(+:stretching_energy)
	for (int i = 0; i < Ele_Num(); i++) {
		ElasticParam& material = materials[material_id[i]];
		real avg_h = 0;

		ArrayF<VectorD, d> vtx;
		MatrixD x_hat;
		for (int j = 0; j < d; j++) {
			vtx[j] = X()[E()[i][j]];
			x_hat.col(j) = vtx[j];
			avg_h += hs[E()[i][j]];
		}
		avg_h *= (real)1 / (real)d;
		real ks = Ks(material.youngs_modulus, avg_h, material.poisson_ratio);

		MatrixD ds;
		NonlinearFemFunc<d>::D(vtx, ds);
		MatrixD deformation = ds * Dm_inv[i];
		MatrixD strain;
		Deformation_To_Strain(deformation, strain);

		stretching_energy += Stretching_Energy(areas_hat[i], ks, material.poisson_ratio, strain);
	}
	return stretching_energy;
}

template<int d> real SoftBodyNonlinearFemThinShell<d>::Total_Bending_Energy() {/*No implementation*/ }

template<> real SoftBodyNonlinearFemThinShell<2>::Total_Bending_Energy() { return 0; /*To be implemented*/ }

template<> real SoftBodyNonlinearFemThinShell<3>::Total_Bending_Energy() {
	real bending_energy = 0;
#pragma omp parallel for reduction(+:bending_energy)
	for (int edge_idx = 0; edge_idx < edges.size(); edge_idx++) {
		ArrayF<int, 4> vtx_idx;
		ArrayF<int, 2> ele_idx;
		if (!Junction_Info(edge_idx, vtx_idx, ele_idx)) {continue;}

		Vector3 n0 = Triangle<3>::Normal(X()[E()[ele_idx[0]][0]], X()[E()[ele_idx[0]][1]], X()[E()[ele_idx[0]][2]]);
		Vector3 n1 = Triangle<3>::Normal(X()[E()[ele_idx[1]][0]], X()[E()[ele_idx[1]][1]], X()[E()[ele_idx[1]][2]]);
		real theta = Dihedral_Angle(n0, n1, X()[vtx_idx[0]], X()[vtx_idx[1]]);

		bending_energy+= Bending_Energy(lambdas[edge_idx], theta, theta_hats[edge_idx]);
	}

	return bending_energy;
}

template<int d> real SoftBodyNonlinearFemThinShell<d>::Lambda(const ArrayF<int, d + 1>& vtx_idx, const ArrayF<int, 2>& ele_idx) {
	real l_hat;
	if constexpr (d == 2) { l_hat = (real)1; }
	else { l_hat = (X0[vtx_idx[0]] - X0[vtx_idx[1]]).norm(); } //May be stored

	real a_hat = areas_hat[ele_idx[0]] + areas_hat[ele_idx[1]];
	ElasticParam& material0 = materials[material_id[ele_idx[0]]];
	ElasticParam& material1 = materials[material_id[ele_idx[1]]];
	real avg_h0, avg_h1, avg_h;
	avg_h0 = avg_h1 = avg_h = (real) 0;
	for (int i = 0; i < d + 1; i++) { avg_h += hs[vtx_idx[i]]; } avg_h /= (real)(d + 1);
	for (int i = 0; i < d; i++) { avg_h0 += hs[E()[ele_idx[0]][i]]; } avg_h0 /= (real)d;
	for (int i = 0; i < d; i++) { avg_h1 += hs[E()[ele_idx[1]][i]]; } avg_h1 /= (real)d;
	real ks0 = Ks(material0.youngs_modulus, avg_h0, material0.poisson_ratio);
	real ks1 = Ks(material1.youngs_modulus, avg_h1, material1.poisson_ratio);
	return Lambda(Kb(avg_h, (real)0.5 * (ks0 + ks1)), a_hat, l_hat);
}

template<> inline real SoftBodyNonlinearFemThinShell<2>::Lambda(real kb, real a_hat, real l_hat) {
	return kb / ((real)4 * a_hat); //the other two is divided in the areas_hat adding together
}

template<> inline real SoftBodyNonlinearFemThinShell<3>::Lambda(real kb, real a_hat, real l_hat) {
	return kb * l_hat * l_hat / ((real)4 * a_hat); //the other two is divided in the areas_hat adding together
}

template<int d> Matrix<real,d> SoftBodyNonlinearFemThinShell<d>::Numerical_Grad_Stretch(int ele_idx, const MatrixD& grad_s) {
	MatrixD grad_s_n;
	const real epsilon=1e-10;
	real stretching_energy= Stretching_Energy(ele_idx);

	ArrayF<VectorD, d> vtx;
	for (int j = 0; j < d; j++) {
		vtx[j] = X()[E()[ele_idx][j]];
	}

	for (int col=0; col<d; col++) {
		for (int row = 0; row < d; row++) {
			real p_tmp=vtx[col][row];
			vtx[col][row]+=epsilon;
			real energy_p=Stretching_Energy(ele_idx);
			grad_s_n(row,col)=(energy_p-stretching_energy)/epsilon;
			vtx[col][row]=p_tmp;
		}
	}

	if (!grad_s_n.isApprox(grad_s, (real)1e-2)) {
		std::cout << "grad_s[" << ele_idx << "]: \n" << grad_s << "\n" << "grad_s_n[" << ele_idx << "]: \n" << grad_s_n << std::endl;
	}
	return grad_s_n;
}

template<int d> Array2DF<Matrix<real, d>, d, d> SoftBodyNonlinearFemThinShell<d>::Numerical_Hess_Stretch(int ele_idx, const MatrixD& grad_s, const MatrixD hess_s[d][d]) {
	Array2DF<MatrixD, d, d> hess_s_n;
	const real epsilon = 1e-5;

	ArrayF<VectorD, d> vtx;
	for (int j = 0; j < d; j++) {
		vtx[j] = X()[E()[ele_idx][j]];
	}

	for (int col = 0; col < d; col++) {
		for (int row = 0; row < d; row++) {
			real p_tmp = vtx[col][row];
			vtx[col][row] += epsilon;
			MatrixD grad_cr;
			Grad_Stretch(ele_idx, grad_cr);
			MatrixD hess_n_cr = (grad_cr - grad_s) / epsilon;
			
			//convert the index from hessian between one element to all other elements to
			//hessian between two vectors
			for (int i = 0; i < d; i++) {
				for (int j = 0; j < d; j++) {
					hess_s_n[j][col](i, row) = hess_n_cr(i,j);
				}
			}

			vtx[col][row] = p_tmp;
		}
	}

	for (int r = 0; r < d; r++) {
		for (int c = 0; c < d; c++) {
			if (!hess_s_n[r][c].isApprox(hess_s[r][c], (real)0.05)) {
				std::cout << "hess_s[" << ele_idx << "]" << std::endl;
				std::cout << "hess_s[" << r << "][" << c << "] \n" << hess_s[r][c] << "\n" << "hess_s_n[" << r << "][" << c << "] \n" << hess_s_n[r][c] << std::endl;

				for (int j = 0; j < d; j++) {
					std::cout << "vtx" << j << std::endl;
					std::cout << X0[E()[ele_idx][j]].transpose() << std::endl;
					std::cout << X()[E()[ele_idx][j]].transpose() << std::endl;
				}
			}
		}
	}

	return hess_s_n;
}

template<int d> Eigen::Matrix<real, d, d + 1> SoftBodyNonlinearFemThinShell<d>::Numerical_Grad_Bend(int jct_idx, ArrayF<int, d + 1>& vtx_idx, ArrayF<int, 2>& ele_idx, const Eigen::Matrix<real, d, d + 1>& grad_b) {}

template<> Eigen::Matrix<real, 2, 3> SoftBodyNonlinearFemThinShell<2>::Numerical_Grad_Bend(int jct_idx, ArrayF<int, 3>& vtx_idx, ArrayF<int, 2>& ele_idx, const Eigen::Matrix<real, 2, 3>& grad_b) { /*Not implemented yet*/ Eigen::Matrix<real, 2, 3> grad_n; return grad_n; }

template<> Eigen::Matrix<real,3,4> SoftBodyNonlinearFemThinShell<3>::Numerical_Grad_Bend(int jct_idx, ArrayF<int, 4>& vtx_idx, ArrayF<int, 2>& ele_idx, const Eigen::Matrix<real, 3, 4>& grad_b)
{
	Eigen::Matrix<real,3,4> grad_b_n;
	real epsilon=1e-10;

	Vector3 n0=Triangle<3>::Normal(X()[E()[ele_idx[0]][0]],X()[E()[ele_idx[0]][1]],X()[E()[ele_idx[0]][2]]);
	Vector3 n1=Triangle<3>::Normal(X()[E()[ele_idx[1]][0]],X()[E()[ele_idx[1]][1]],X()[E()[ele_idx[1]][2]]);
	real theta=Dihedral_Angle(n0,n1,X()[vtx_idx[0]],X()[vtx_idx[1]]);
	real bending_energy=Bending_Energy(lambdas[jct_idx], theta, theta_hats[jct_idx]);

	for (int col=0; col<4; col++) {
		for (int row = 0; row < 3; row++) {
			real p_tmp=X()[vtx_idx[col]][row];
			X()[vtx_idx[col]][row]+=epsilon;
			n0=Triangle<3>::Normal(X()[E()[ele_idx[0]][0]],X()[E()[ele_idx[0]][1]],X()[E()[ele_idx[0]][2]]);
			n1=Triangle<3>::Normal(X()[E()[ele_idx[1]][0]],X()[E()[ele_idx[1]][1]],X()[E()[ele_idx[1]][2]]);
			theta=Dihedral_Angle(n0,n1,X()[vtx_idx[0]],X()[vtx_idx[1]]);
			real energy_p=Bending_Energy(lambdas[jct_idx], theta, theta_hats[jct_idx]);
			grad_b_n(row,col)=(energy_p-bending_energy)/epsilon;
			X()[vtx_idx[col]][row]=p_tmp;
		}
	}
	if (!grad_b_n.isApprox(grad_b, (real)1e-2)) {
		std::cout << "grad_b[" << jct_idx << "]: \n" << grad_b << "\n" << "grad_b_n[" << jct_idx << "]: \n" << grad_b_n << std::endl;
	}
	return grad_b_n;
}


template<int d> Array2DF<Matrix<real,d>, d + 1, d + 1> SoftBodyNonlinearFemThinShell<d>::Numerical_Hess_Bend(int jct_idx, const ArrayF<int, d + 1>& vtx_idx, const ArrayF<int, 2>& ele_idx, const Eigen::Matrix<real, d, d + 1>& grad_b, const MatrixD hess_b[d + 1][d + 1]) {
	Array2DF<MatrixD, d + 1, d + 1> hess_b_n;
	const real epsilon = 1e-10;

	for (int col = 0; col < d+1; col++) {
		for (int row = 0; row < d; row++) {
			real p_tmp = X()[vtx_idx[col]][row];
			X()[vtx_idx[col]][row] += epsilon;

			Eigen::Matrix<real, d, d + 1> grad_cr;
			Grad_Bend(jct_idx, grad_cr, vtx_idx, ele_idx);
			
			Eigen::Matrix<real, d, d + 1> hess_n_cr = (grad_cr - grad_b) / epsilon;

			//convert the index from hessian between one element to all other elements to
			//hessian between two vectors
			for (int i = 0; i < d; i++) {
				for (int j = 0; j < d+1; j++) {
					hess_b_n[j][col](i, row) = hess_n_cr(i, j);
				}
			}

			X()[vtx_idx[col]][row] = p_tmp;
		}
	}

	for (int r = 0; r < d + 1; r++) {
		for (int c = 0; c < d + 1; c++) {
			if (!hess_b_n[r][c].isApprox(hess_b[r][c], (real)1e-2)) {
				std::cout << "jct_idx: " << jct_idx << std::endl;
				std::cout << "hess_b[" << r << "][" << c << "] \n" << hess_b[r][c] << "\n" << "hess_b_n[" << r << "][" << c << "] \n" << hess_b_n[r][c] << std::endl;
			}
		}
	}
	return hess_b_n;
}

template<int d> Array2DF<Matrix<real, d>, d + 1, d + 1>  SoftBodyNonlinearFemThinShell<d>::Numerical_Hess_Theta(const ArrayF<int, d + 1>& vtx_idx,const ArrayF<int, 2>& ele_idx, const Eigen::Matrix<real, d, d + 1>& grad_theta) {
	Array2DF<MatrixD, d + 1, d + 1> hess_n;
	const real epsilon = 1e-10;

	for (int col = 0; col < d + 1; col++) {
		for (int row = 0; row < d; row++) {
			real p_tmp = X()[vtx_idx[col]][row];
			X()[vtx_idx[col]][row] += epsilon;

			Eigen::Matrix<real, d, d + 1> grad_cr;
			Grad_Theta(grad_cr, vtx_idx, ele_idx);

			Eigen::Matrix<real, d, d + 1> hess_n_cr = (grad_cr - grad_theta) / epsilon;

			//convert the index from hessian between one element to all other elements to
			//hessian between two vectors
			for (int i = 0; i < d; i++) {
				for (int j = 0; j < d + 1; j++) {
					hess_n[j][col](i, row) = hess_n_cr(i, j);
				}
			}

			X()[vtx_idx[col]][row] = p_tmp;
		}
	}

	return hess_n;
}

template<class T_ARRAY> int Element_Edges(const Vector2i& v,T_ARRAY& edges)
{edges[0][0]=v[0];edges[1][0]=v[1];return 2;}

template<class T_ARRAY> int Element_Edges(const Vector3i& v,T_ARRAY& edges)
{edges[0]=Vector2i(v[0],v[1]);edges[1]=Vector2i(v[1],v[2]);edges[2]=Vector2i(v[2],v[0]);return 3;}



void Grad_Q(const ArrayF<Vector3, 3>& vtx, const int i, const int j, const Vector3& ps, const real& qs_i, const ArrayF<Vector3, 3>& ls, const real& a, Vector3& grad_q) {
	int jr = (j + 1) % 3;
	int jl = (j + 2) % 3;

	real tmp1 = ps[jr] * qs_i / (real)8 / a / a;
	if (i == jr) { tmp1 -= (real)0.5 / (a * ls[i].norm()); }

	real tmp2 = ps[jl] * qs_i / (real)8 / a / a;
	if (i == jl) { tmp2 -= (real)0.5 / (a * ls[i].norm()); }

	grad_q = tmp1 * ls[jr] - tmp2 * ls[jl];
}

void Grad_R(const ArrayF<Vector3, 3>& vtx, const int i, const int j, const Vector3& ps, const ArrayF<Vector3, 3>& ls, const real& a, ArrayF<Vector3, 2>& grad_r) {
	int jr = (j + 1) % 3;
	int jl = (j + 2) % 3;

	Vector3 grad_a = (real)1 / (real)8 / a * (ps[jl] * ls[jl] - ps[jr] * ls[jr]);
	Vector3 grad_l = ((int)(i == jl)) / ls[jl].norm() * ls[jl] - ((int)(i == jr)) / ls[jr].norm() * ls[jr];

	Vector3 coef2 = (grad_a * ls[i].norm() + grad_l * a) / ((real)4 * a * a * ls[i].dot(ls[i]));
	real coef1 = a * ls[i].norm() * (real)2;

	int i_p = (i + 2) % 3;
	Vector3 tmp1 = ((((int)(i_p == jr)) + ((int)(i_p == j)) - ((int)(i_p == jl))) * ls[jl] + (((int)(i_p == jr)) - ((int)(i_p == j)) - ((int)(i_p == jl))) * ls[jr]);
	grad_r[0] = tmp1 / coef1 - ps[i_p] * coef2;

	i_p = (i + 1) % 3;
	tmp1 = ((((int)(i_p == jr)) + ((int)(i_p == j)) - ((int)(i_p == jl))) * ls[jl] + (((int)(i_p == jr)) - ((int)(i_p == j)) - ((int)(i_p == jl))) * ls[jr]);
	grad_r[1] = tmp1 / coef1 - ps[i_p] * coef2;
}

void Grad_N(const ArrayF<Vector3, 3>& vtx, const ArrayF<Vector3, 3>& ls, const real& a, ArrayF<Matrix3, 3>& grad_n) {
	Vector3 n = Triangle<3>::Normal(vtx[0], vtx[1], vtx[2]);
	for (int i = 0; i < 3; i++) {
		grad_n[i] = -n * ((real)0.5 * ls[i].cross(n) / a).transpose();
	}
}

template<int d> bool SoftBodyNonlinearFemThinShell<d>::Junction_Info(int edge_idx, ArrayF<int, d + 1>& vtx_idx, ArrayF<int, 2>& ele_idx)
{
	if constexpr (d == 3) {
		vtx_idx[0] = edges[edge_idx][0];
		vtx_idx[1] = edges[edge_idx][1];
		Array<int> incident_elements;
		Value_Array(edge_element_hashtable, edges[edge_idx], incident_elements);
		if (incident_elements.size() == 2) { //shared edge
			ele_idx[0] = incident_elements[0], ele_idx[1] = incident_elements[1];
			vtx_idx[2] = Third_Vertex(vtx_idx[0], vtx_idx[1], E()[ele_idx[0]]);
			vtx_idx[3] = Third_Vertex(vtx_idx[0], vtx_idx[1], E()[ele_idx[1]]);

			for (int i = 0; i < 3; i++) {
				if (E()[ele_idx[0]][i] == vtx_idx[2]) { vtx_idx[0] = E()[ele_idx[0]][(i + 1) % 3]; vtx_idx[1] = E()[ele_idx[0]][(i + 2) % 3]; }
			}
			return true;
		}
		else {
			return false;
		}
	}
	else if constexpr (d == 2) {
		//need to be implemented
		//vtx_idx[0] = edges[edge_idx][0];
		//vtx_idx[1] = edges[edge_idx][1];
		//Array<int> incident_elements;
		//Value_Array(edge_element_hashtable, edges[edge_idx], incident_elements);
		//if (incident_elements.size() == 2) { //shared edge
		//	ele_idx[0] = incident_elements[0], ele_idx[1] = incident_elements[1];
		//	vtx_idx[2] = Opposite_Vertex(vtx_idx[0], vtx_idx[1], E()[ele_idx[0]]);
		//	vtx_idx[3] = Opposite_Vertex(vtx_idx[0], vtx_idx[1], E()[ele_idx[1]]);

		//	for (int i = 0; i < 3; i++) {
		//		if (E()[ele_idx[0]][i] == vtx_idx[2]) { vtx_idx[0] = E()[ele_idx[0]][(i + 1) % 3]; vtx_idx[1] = E()[ele_idx[0]][(i + 2) % 3]; }
		//	}
		//	return true;
		//}
		//else {
		//	return false;
		//}
		return false;
	}
}

////////////////////////////////////////////////////////////////////////
//omp accelerated functions

inline void Add_Element_Force_To_Vertices(Array<Vector2>& F,const Vector3i& e,const Matrix2& ff)
{
	Vector2& f0=F[e[0]];
	Vector2& f1=F[e[1]];
	Vector2& f2=F[e[2]];
	Vector2 c0=ff.col(0);
	Vector2 c1=ff.col(1);
	Vector2 c2=c0+c1;
#pragma omp atomic
	f0[0]-=c0[0];
#pragma omp atomic
	f0[1]-=c0[1];
#pragma omp atomic
	f1[0]-=c1[0];
#pragma omp atomic
	f1[1]-=c1[1];
#pragma omp atomic
	f2[0]+=c2[0];
#pragma omp atomic
	f2[1]+=c2[1];
}

inline void Add_Element_Force_To_Vertices(Array<Vector3>& F,const Vector4i& e,const Matrix3& ff)
{ 
	Vector3& f0=F[e[0]];
	Vector3& f1=F[e[1]];
	Vector3& f2=F[e[2]];
	Vector3& f3=F[e[3]];
	Vector3 c0=ff.col(0);
	Vector3 c1=ff.col(1);
	Vector3 c2=ff.col(2);
	Vector3 c3=c0+c1+c2;
#pragma omp atomic
	f0[0]-=c0[0];
#pragma omp atomic
	f0[1]-=c0[1];
#pragma omp atomic
	f0[2]-=c0[2];
#pragma omp atomic
	f1[0]-=c1[0];
#pragma omp atomic
	f1[1]-=c1[1];
#pragma omp atomic
	f1[2]-=c1[2];
#pragma omp atomic
	f2[0]-=c2[0];
#pragma omp atomic
	f2[1]-=c2[1];
#pragma omp atomic
	f2[2]-=c2[2];
#pragma omp atomic
	f3[0]+=c3[0];
#pragma omp atomic
	f3[1]+=c3[1];
#pragma omp atomic
	f3[2]+=c3[2];
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Add_Block_Helper(SparseMatrix<real>& K, const int i, const int j, const MatrixD& Ks)
{
	SparseFunc::Add_Block<d, MatrixD>(K, i, j, Ks);
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Set_Block(VectorX& b, const int i, const VectorD& bi)
{
	for (int ii = 0; ii < d; ii++)b[i * d + ii] = bi[ii];
}

template<int d> void SoftBodyNonlinearFemThinShell<d>::Add_Block(VectorX& b, const int i, const VectorD& bi)
{
	for (int ii = 0; ii < d; ii++)b[i * d + ii] += bi[ii];
}

template class SoftBodyNonlinearFemThinShell<2>;
template class SoftBodyNonlinearFemThinShell<3>;
